#include "convex_hull_general.h"
#include "convex_hull_serial.h"
#include <GL/glut.h>
#include <algorithm>
#include <chrono>
#include <cmath>
#include <cstdlib>
#include <cstring>
#include <ctime>
#include <hip/hip_runtime.h>
#include <>
#include <iostream>
#include <set>
#include <thread>
#include <thrust/host_vector.h>
#include <thrust/reduce.h>

#define htkCheck(stmt)                                                                             \
  do                                                                                               \
  {                                                                                                \
    hipError_t err = stmt;                                                                        \
    if (err != hipSuccess)                                                                        \
    {                                                                                              \
      std::cerr << "Failed to run stmt: " << #stmt << std::endl;                                   \
      std::cerr << "Got CUDA error (" << err << "): " << hipGetErrorString(err) << std::endl;     \
      std::cerr << "File: " << __FILE__ << ", Line: " << __LINE__ << std::endl;                    \
      exit(1);                                                                                     \
    }                                                                                              \
  } while (0)

using namespace std;

vector<Point> global_points;
vector<Point> global_hull;

void display()
{
  glClear(GL_COLOR_BUFFER_BIT);
  glColor3f(1.0, 1.0, 1.0);

  // Draw all points
  glBegin(GL_POINTS);
  for (const auto &point : global_points)
  {
    glVertex2f(point.x, point.y);
  }
  glEnd();

  // Draw the hull
  glColor3f(1.0, 0.0, 0.0);
  glBegin(GL_LINE_LOOP);
  for (const auto &point : global_hull)
  {
    glVertex2f(point.x, point.y);
  }
  glEnd();

  glutSwapBuffers();
}

// Timer function for real-time updates
void timer(int)
{
  // Re-render the display
  glutPostRedisplay();

  // Add a delay for visualization purposes
  this_thread::sleep_for(chrono::milliseconds(100));

  // Register the timer callback again
  glutTimerFunc(100, timer, 0);
}

// Initialize OpenGL
void initOpenGL()
{
  glClearColor(0.0, 0.0, 0.0, 1.0);
  glMatrixMode(GL_PROJECTION);
  glLoadIdentity();
  gluOrtho2D(0.0, 1000.0, 0.0, 1000.0);
}

int main(int argc, char *argv[])
{
  Config config;
  int exit_status = parse_args(argc, argv, &config);
  if (exit_status != 0)
  {
    return 1;
  }

  cout << config.num_points << endl;
  cout << config.command << endl;
  int case_id;
  if (strcmp(config.command, "both") == 0)
    case_id = 0;
  else if (strcmp(config.command, "serial") == 0)
    case_id = 1;
  else if (strcmp(config.command, "parallel") == 0)
    case_id = 2;
  else
  {
    cerr << "Invalid command" << endl;
    return 1;
  }

  cout << "case_id: " << case_id << endl;

  global_points = generate_random_points(config.num_points);

  cout << "Generated points" << endl;

  // Sort points by x value, y value is tie breaker
  sort(global_points.begin(), global_points.end(),
       [](const Point &a, const Point &b) { return (a.x < b.x) || (a.x == b.x && a.y < b.y); });

  // Initialize OpenGL
  glutInit(&argc, argv);
  glutInitDisplayMode(GLUT_DOUBLE | GLUT_RGB);
  glutInitWindowSize(800, 800);
  glutCreateWindow("Convex Hull Visualization");

  initOpenGL();
  cout << "Window initialized" << endl;

  vector<Point> serial_hull, parallel_hull;
  std::chrono::time_point<std::chrono::high_resolution_clock> serial_start, serial_end,
      parallel_start, parallel_end;
  double serial_time, parallel_time;
  int hullCount;

  cout << "Variables needed for case statements declared" << endl;

  // NOTE: WHERE I LEFT OFF: Implement the switch statment logic for the command
  switch (case_id)
  {
  case 0: // both case
    serial_start = std::chrono::high_resolution_clock::now();
    serial_hull = divide(global_points);
    serial_end = std::chrono::high_resolution_clock::now();
    serial_time = std::chrono::duration<double>(serial_end - serial_start).count();
    break;
  case 1: // serial case
    serial_start = std::chrono::high_resolution_clock::now();
    serial_hull = divide(global_points);
    serial_end = std::chrono::high_resolution_clock::now();
    serial_time = std::chrono::duration<double>(serial_end - serial_start).count();
    global_hull = serial_hull;
    break;
  case 2: // parallel case
    hullCount = 0;
    parallel_start = std::chrono::high_resolution_clock::now();
    parallel_hull.resize(global_points.size());
    cout << "hull size resized properlly" << endl;
    divide_kernel_caller(global_points.data(), global_points.size(), global_points[0],
                         global_points.back(), parallel_hull.data(), &hullCount);
    cout << "divided kernel caller returned" << endl;
    parallel_end = std::chrono::high_resolution_clock::now();
    parallel_time = std::chrono::duration<double>(parallel_end - parallel_start).count();
    global_hull = parallel_hull;
    break;
  default:
    break;
  }
  glutDisplayFunc(display);
  glutTimerFunc(0, timer, 0);
  cout << "Drawing" << endl;
  // Start the main loop
  glutMainLoop();
  return 0;
}
